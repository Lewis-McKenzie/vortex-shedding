#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <fcntl.h>
#include <math.h>
#include <time.h>

#include "data.cuh"
#include "vtk.cuh"
#include "setup.cuh"
#include "boundary.cuh"
#include "args.cuh"

struct timespec timer;
double get_time() {
	clock_gettime(CLOCK_MONOTONIC, &timer); 
	return (double) (timer.tv_sec + timer.tv_nsec / 1000000000.0);
}

#define time(func, timer) if(print_time && threadIdx.x == 0){timer = get_time();func;timer = get_time() - timer;}else{func;}

#define print_timer(name, timer) if(print_time)printf("%s: %lf\n", name, timer);

#define init_outer_loop(i, limit, addon) i = threadIdx.x * (imax+2) / blockDim.x;limit = (threadIdx.x+1) * (imax+2) / blockDim.x;if (i == 0) {i = 1;} else if (limit > imax+addon) {i_end = imax+addon;}

#define debug_cuda(i, limit) printf("thread: %d out of %d on block %d. start: %d end: %d\n", threadIdx.x, blockDim.x, blockIdx.x, i, limit);

__device__ double block_reduce_sum(double value, double *reduction_buffer) {
    int tid = threadIdx.x;
    reduction_buffer[tid] = value;
    __syncthreads();
    
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s)
            reduction_buffer[tid] += reduction_buffer[tid + s];

        __syncthreads();
    }

    return reduction_buffer[0];
}

__global__ void block_reduce_sum_buffer(double *reduction_buffer) {
    int tid = threadIdx.x;
    __syncthreads();
    
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s)
            reduction_buffer[tid] += reduction_buffer[tid + s];

        __syncthreads();
    }
}



/**
 * @brief Computation of tentative velocity field (f, g)
 * 
 */
__global__ void compute_tentative_velocity(double** u, double **v, char **flag, double **f, double **g, int imax, int jmax, double del_t) {
    int i, i_end;
    init_outer_loop(i, i_end, 0);
    //debug_cuda(i, i_end);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax+1; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i+1][j] & C_F)) {
                double du2dx = ((u[i][j] + u[i+1][j]) * (u[i][j] + u[i+1][j]) +
                                y * fabs(u[i][j] + u[i+1][j]) * (u[i][j] - u[i+1][j]) -
                                (u[i-1][j] + u[i][j]) * (u[i-1][j] + u[i][j]) -
                                y * fabs(u[i-1][j] + u[i][j]) * (u[i-1][j]-u[i][j]))
                                / (4.0 * delx);
                double duvdy = ((v[i][j] + v[i+1][j]) * (u[i][j] + u[i][j+1]) +
                                y * fabs(v[i][j] + v[i+1][j]) * (u[i][j] - u[i][j+1]) -
                                (v[i][j-1] + v[i+1][j-1]) * (u[i][j-1] + u[i][j]) -
                                y * fabs(v[i][j-1] + v[i+1][j-1]) * (u[i][j-1] - u[i][j]))
                                / (4.0 * dely);
                double laplu = (u[i+1][j] - 2.0 * u[i][j] + u[i-1][j]) / delx / delx +
                                (u[i][j+1] - 2.0 * u[i][j] + u[i][j-1]) / dely / dely;
   
                f[i][j] = u[i][j] + del_t * (laplu / Re - du2dx - duvdy);
            } else {
                f[i][j] = u[i][j];
            }
        }
    }

    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i][j+1] & C_F)) {
                double duvdx = ((u[i][j] + u[i][j+1]) * (v[i][j] + v[i+1][j]) +
                                y * fabs(u[i][j] + u[i][j+1]) * (v[i][j] - v[i+1][j]) -
                                (u[i-1][j] + u[i-1][j+1]) * (v[i-1][j] + v[i][j]) -
                                y * fabs(u[i-1][j] + u[i-1][j+1]) * (v[i-1][j]-v[i][j]))
                                / (4.0 * delx);
                double dv2dy = ((v[i][j] + v[i][j+1]) * (v[i][j] + v[i][j+1]) +
                                y * fabs(v[i][j] + v[i][j+1]) * (v[i][j] - v[i][j+1]) -
                                (v[i][j-1] + v[i][j]) * (v[i][j-1] + v[i][j]) -
                                y * fabs(v[i][j-1] + v[i][j]) * (v[i][j-1] - v[i][j]))
                                / (4.0 * dely);
                double laplv = (v[i+1][j] - 2.0 * v[i][j] + v[i-1][j]) / delx / delx +
                                (v[i][j+1] - 2.0 * v[i][j] + v[i][j-1]) / dely / dely;

                g[i][j] = v[i][j] + del_t * (laplv / Re - duvdx - dv2dy);
            } else {
                g[i][j] = v[i][j];
            }
        }
    }

    /* f & g at external boundaries */
    if (threadIdx.x == 0) {
        for (int j = 1; j < jmax+1; j++) {
            f[0][j]    = u[0][j];
        }
    } else if (threadIdx.x == blockDim.x - 1) {
        for (int j = 1; j < jmax+1; j++) {
            f[imax][j] = u[imax][j];
        }
    }
    init_outer_loop(i, i_end, 1);
    for (i; i < i_end; i++) {
        g[i][0]    = v[i][0];
        g[i][jmax] = v[i][jmax];
    }
}


/**
 * @brief Calculate the right hand side of the pressure equation 
 * 
 */
__global__ void compute_rhs(char **flag, double **f, double **g, double **rhs, int imax, int jmax, double del_t) {
    int i, i_end;
    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1;j < jmax+1; j++) {
            if (flag[i][j] & C_F) {
                /* only for fluid and non-surface cells */
                rhs[i][j] = ((f[i][j] - f[i-1][j]) / delx + 
                             (g[i][j] - g[i][j-1]) / dely)
                            / del_t;
            }
        }
    }
}


__global__ void init_p0(double **p, char **flag, int imax, int jmax, double *reduction_buffer) {
    double p0 = 0.0;
    /* Calculate sum of squares */
    int i, i_end;
    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax+1; j++) {
            if (flag[i][j] & C_F) { p0 += p[i][j] * p[i][j]; }
        }
    }
    reduction_buffer[threadIdx.x] = p0;
}


__global__ void update_p(double **p, char **flag, double **rhs, int imax, int jmax) {
    int i, i_end;
    for (int rb = 0; rb < 2; rb++) {

        init_outer_loop(i, i_end, 1);
        for (; i < i_end; i++) {
            for (int j = 1; j < jmax+1; j++) {
                if ((i + j) % 2 != rb) { continue; }
                if (flag[i][j] == (C_F | B_NSEW)) {
                    /* five point star for interior fluid cells */
                    p[i][j] = (1.0 - omega) * p[i][j] - 
                            beta_2 * ((p[i+1][j] + p[i-1][j] ) * rdx2
                                        + (p[i][j+1] + p[i][j-1]) * rdy2
                                        - rhs[i][j]);
                } else if (flag[i][j] & C_F) { 
                    /* modified star near boundary */

                    double eps_E = ((flag[i+1][j] & C_F) ? 1.0 : 0.0);
                    double eps_W = ((flag[i-1][j] & C_F) ? 1.0 : 0.0);
                    double eps_N = ((flag[i][j+1] & C_F) ? 1.0 : 0.0);
                    double eps_S = ((flag[i][j-1] & C_F) ? 1.0 : 0.0);

                    double beta_mod = -omega / ((eps_E + eps_W) * rdx2 + (eps_N + eps_S) * rdy2);
                    p[i][j] = (1.0 - omega) * p[i][j] -
                        beta_mod * ((eps_E * p[i+1][j] + eps_W * p[i-1][j]) * rdx2
                                        + (eps_N * p[i][j+1] + eps_S * p[i][j-1]) * rdy2
                                        - rhs[i][j]);
                }
            }
        }
    }

}

__global__ void update_res(double **p, char **flag, double **rhs, int imax, int jmax, double res, double *reduction_buffer) {
    /* computation of residual */
    int i, i_end;
    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax+1; j++) {
            if (flag[i][j] & C_F) {
                double eps_E = ((flag[i+1][j] & C_F) ? 1.0 : 0.0);
                double eps_W = ((flag[i-1][j] & C_F) ? 1.0 : 0.0);
                double eps_N = ((flag[i][j+1] & C_F) ? 1.0 : 0.0);
                double eps_S = ((flag[i][j-1] & C_F) ? 1.0 : 0.0);

                /* only fluid cells */
                double add = (eps_E * (p[i+1][j] - p[i][j]) - 
                    eps_W * (p[i][j] - p[i-1][j])) * rdx2  +
                    (eps_N * (p[i][j+1] - p[i][j]) -
                        eps_S * (p[i][j] - p[i][j-1])) * rdy2  -  rhs[i][j];
                res += add * add;
            }
        }
    }
    reduction_buffer[threadIdx.x] = res;
}

/**
 * @brief Red/Black SOR to solve the poisson equation.
 * 
 * @return Calculated residual of the computation
 * 
 */
double poisson() {

    init_p0<<<grid_dim, block_dim>>>(p, flag, imax, jmax, reduction_buffer);
    block_reduce_sum_buffer<<<grid_dim, block_dim>>>(reduction_buffer);
   
    double p0 = sqrt(reduction_buffer[0] / fluid_cells); 
    if (p0 < 0.0001) { p0 = 1.0; }

    /* Red/Black SOR-iteration */
    int iter;
    double res = 0.0;
    for (iter = 0; iter < itermax; iter++) {

        update_p<<<grid_dim, block_dim>>>(p, flag, rhs, imax, jmax);
        update_res<<<grid_dim, block_dim>>>(p, flag, rhs, imax, jmax, res, reduction_buffer);
        block_reduce_sum_buffer<<<grid_dim, block_dim>>>(reduction_buffer);
        hipDeviceSynchronize();

        res = sqrt(reduction_buffer[0] / fluid_cells) / p0;
        
        /* convergence? */
        if (res < eps) break;
    }

    return res;
}


/**
 * @brief Update the velocity values based on the tentative
 * velocity values and the new pressure matrix
 */
__global__ void update_velocity(double **u, double **v, double **p, char ** flag, double **f, double **g, int imax, int jmax, double del_t) {
    int i, i_end;
    init_outer_loop(i, i_end, -2);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax-1; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i+1][j] & C_F)) {
                u[i][j] = f[i][j] - (p[i+1][j] - p[i][j]) * del_t / delx;
            }
        }
    }
    
    init_outer_loop(i, i_end, -1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax-2; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i][j+1] & C_F)) {
                v[i][j] = g[i][j] - (p[i][j+1] - p[i][j]) * del_t / dely;
            }
        }
    }
}


/**
 * @brief Set the timestep size so that we satisfy the Courant-Friedrichs-Lewy
 * conditions. Otherwise the simulation becomes unstable.
 */
void set_timestep_interval() {
    /* del_t satisfying CFL conditions */
    if (tau >= 1.0e-10) { /* else no time stepsize control */
        double umax = 1.0e-10;
        double vmax = 1.0e-10; 
        
        for (int i = 0; i < imax+2; i++) {
            for (int j = 1; j < jmax+2; j++) {
                umax = fmax(fabs(u[i][j]), umax);
            }
        }

        for (int i = 1; i < imax+2; i++) {
            for (int j = 0; j < jmax+2; j++) {
                vmax = fmax(fabs(v[i][j]), vmax);
            }
        }

        double deltu = delx / umax;
        double deltv = dely / vmax; 
        double deltRe = 1.0 / (1.0 / (delx * delx) + 1 / (dely * dely)) * Re / 2.0;

        if (deltu < deltv) {
            del_t = fmin(deltu, deltRe);
        } else {
            del_t = fmin(deltv, deltRe);
        }
        del_t = tau * del_t; /* multiply by safety factor */
    }
}


void main_loop() {
    double res, t, ten_t, rhs_t, pois_t, vel_t, bound_t;

    apply_boundary_conditions<<<grid_dim, block_dim>>>(u, v, flag, imax, jmax);

    /* Main loop */
    int iters = 0;
    for (t = 0.0; t < t_end; t += del_t, iters++) {
        if (!fixed_dt) {
            set_timestep_interval();
        }

        (compute_tentative_velocity<<<grid_dim, block_dim>>>(u, v, flag, f, g, imax, jmax, del_t), ten_t);

        (compute_rhs<<<grid_dim, block_dim>>>(flag, f, g, rhs, imax, jmax, del_t), rhs_t);

        (res = poisson(), pois_t);

        (update_velocity<<<grid_dim, block_dim>>>(u, v, p, flag, f, g, imax, jmax, del_t), vel_t);

        (apply_boundary_conditions<<<grid_dim, block_dim>>>(u, v, flag, imax, jmax), bound_t);

        if ((iters % output_freq == 0)) {
            hipDeviceSynchronize();
            printf("Step %8d, Time: %14.8e (del_t: %14.8e), Residual: %14.8e\n", iters, t+del_t, del_t, res);

            if ((!no_output) && (enable_checkpoints)) {
                write_checkpoint(iters, t+del_t);
            }
        }
    } /* End of main loop */

    printf("Step %8d, Time: %14.8e, Residual: %14.8e\n", iters, t, res);
    printf("Simulation complete.\n");

    if (!no_output) {
        write_result(iters, t);
    }
}


/**
 * @brief The main routine that sets up the problem and executes the solving routines routines
 * 
 * @param argc The number of arguments passed to the program
 * @param argv An array of the arguments passed to the program
 * @return int The return value of the application
 */
int main(int argc, char *argv[]) {
    double setup_time;

    setup_time = get_time();
    set_defaults();
    parse_args(argc, argv);
    setup<<<1, 1>>>(imax, jmax);
    hipDeviceSynchronize();

    if (verbose) print_opts();

    allocate_arrays();
    problem_set_up();
    setup_time = get_time() - setup_time;
    print_timer("Setup", setup_time);

    main_loop();


    free_arrays();

    return 0;
}
