#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <fcntl.h>
#include <math.h>
#include <time.h>

#include "data.cuh"
#include "vtk.cuh"
#include "setup.cuh"
#include "boundary.cuh"
#include "args.cuh"

// loop between 1 and imax+1
#define init_outer_loop(i, limit) {int i_block_start = blockIdx.x * (imax / gridDim.x);i = i_block_start + threadIdx.x * (imax / gridDim.x) / blockDim.x + 1;limit = max(i+1, i_block_start + (threadIdx.x + 1)  * (imax / gridDim.x) / blockDim.x+1);}

// loop between 1 and jmax+1
#define init_inner_loop(j, limit) {if (blockDim.x > imax / gridDim.x) {int threads = blockDim.x / (imax / gridDim.x);int iters = jmax / threads;j = (threadIdx.x % threads) * iters + 1;limit = ((threadIdx.x % threads) + 1) * iters + 1;}else {j = 1;limit = jmax+1;}}

#define debug_cuda(i, limit) printf("thread: %d out of %d on block %d. start: %d end: %d\n", threadIdx.x, blockDim.x, blockIdx.x, i, limit);

__global__ void block_reduce_sum_buffer(double *reduction_buffer) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s)
            reduction_buffer[tid] += reduction_buffer[tid + s];

        __syncthreads();
    }
}

__global__ void grid_reduce_sum_buffer(double *reduction_buffer, int block_dim) {
    // index is the start of each block buffer
    int tid = threadIdx.x * block_dim;

    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s)
            reduction_buffer[tid] += reduction_buffer[tid + s*block_dim];

        __syncthreads();
    }
}

double reduce(double *reduction_buffer) {
    block_reduce_sum_buffer<<<grid_dim, block_dim>>>(reduction_buffer);
    grid_reduce_sum_buffer<<<1, grid_dim>>>(reduction_buffer, block_dim);
    checkCuda(hipDeviceSynchronize());
    return reduction_buffer[0];
}


/**
 * @brief Computation of tentative velocity field (f, g)
 * 
 */
__global__ void compute_tentative_velocity(double** u, double **v, char **flag, double **f, double **g, int imax, int jmax, double del_t, double Re, double delx, double dely) {

    int i_start, i_end, j_start, j_end;
    init_outer_loop(i_start, i_end);
    init_inner_loop(j_start, j_end);
    for (int i = i_start; i < i_end && i < imax; i++) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i+1][j] & C_F)) {
                double du2dx = ((u[i][j] + u[i+1][j]) * (u[i][j] + u[i+1][j]) +
                                y * fabs(u[i][j] + u[i+1][j]) * (u[i][j] - u[i+1][j]) -
                                (u[i-1][j] + u[i][j]) * (u[i-1][j] + u[i][j]) -
                                y * fabs(u[i-1][j] + u[i][j]) * (u[i-1][j]-u[i][j]))
                                / (4.0 * delx);
                double duvdy = ((v[i][j] + v[i+1][j]) * (u[i][j] + u[i][j+1]) +
                                y * fabs(v[i][j] + v[i+1][j]) * (u[i][j] - u[i][j+1]) -
                                (v[i][j-1] + v[i+1][j-1]) * (u[i][j-1] + u[i][j]) -
                                y * fabs(v[i][j-1] + v[i+1][j-1]) * (u[i][j-1] - u[i][j]))
                                / (4.0 * dely);
                double laplu = (u[i+1][j] - 2.0 * u[i][j] + u[i-1][j]) / delx / delx +
                                (u[i][j+1] - 2.0 * u[i][j] + u[i][j-1]) / dely / dely;
   
                f[i][j] = u[i][j] + del_t * (laplu / Re - du2dx - duvdy);
            } else {
                f[i][j] = u[i][j];
            }
        }
    }

    for (int i = i_start; i < i_end && i < imax+1; i++) {
        for (int j = j_start; j < j_end && j < jmax; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i][j+1] & C_F)) {
                double duvdx = ((u[i][j] + u[i][j+1]) * (v[i][j] + v[i+1][j]) +
                                y * fabs(u[i][j] + u[i][j+1]) * (v[i][j] - v[i+1][j]) -
                                (u[i-1][j] + u[i-1][j+1]) * (v[i-1][j] + v[i][j]) -
                                y * fabs(u[i-1][j] + u[i-1][j+1]) * (v[i-1][j]-v[i][j]))
                                / (4.0 * delx);
                double dv2dy = ((v[i][j] + v[i][j+1]) * (v[i][j] + v[i][j+1]) +
                                y * fabs(v[i][j] + v[i][j+1]) * (v[i][j] - v[i][j+1]) -
                                (v[i][j-1] + v[i][j]) * (v[i][j-1] + v[i][j]) -
                                y * fabs(v[i][j-1] + v[i][j]) * (v[i][j-1] - v[i][j]))
                                / (4.0 * dely);
                double laplv = (v[i+1][j] - 2.0 * v[i][j] + v[i-1][j]) / delx / delx +
                                (v[i][j+1] - 2.0 * v[i][j] + v[i][j-1]) / dely / dely;

                g[i][j] = v[i][j] + del_t * (laplv / Re - duvdx - dv2dy);
            } else {
                g[i][j] = v[i][j];
            }
        }
    }

    /* f & g at external boundaries */
    if (i_start == 1) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            f[0][j]    = u[0][j];
        }
    } 
    if (i_end == imax+1) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            f[imax][j] = u[imax][j];
        }
    }
    if (j_start == 1) {
        for (int i = i_start; i < i_end && i < imax+1; i++) {
            g[i][0]    = v[i][0];
        }
    }
    if (j_end == jmax+1) {
        for (int i = i_start; i < i_end && i < imax+1; i++) {
            g[i][jmax] = v[i][jmax];
        }
    }
}


/**
 * @brief Calculate the right hand side of the pressure equation 
 * 
 */
__global__ void compute_rhs(char **flag, double **f, double **g, double **rhs, int imax, int jmax, double del_t, double delx, double dely) {
    int i_start, i_end, j_start, j_end;
    init_outer_loop(i_start, i_end);
    init_inner_loop(j_start, j_end);
    for (int i = i_start; i < i_end && i < imax+1; i++) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            if (flag[i][j] & C_F) {
                /* only for fluid and non-surface cells */
                rhs[i][j] = ((f[i][j] - f[i-1][j]) / delx + 
                             (g[i][j] - g[i][j-1]) / dely)
                            / del_t;
            }
        }
    }
}


__global__ void init_p0(double **p, char **flag, int imax, int jmax, double *reduction_buffer) {
    double p0 = 0.0;
    /* Calculate sum of squares */
    int i_start, i_end, j_start, j_end;
    init_outer_loop(i_start, i_end);
    init_inner_loop(j_start, j_end);
    for (int i = i_start; i < i_end && i < imax+1; i++) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            if (flag[i][j] & C_F) { p0 += p[i][j] * p[i][j]; }
        }
    }
    reduction_buffer[blockIdx.x * blockDim.x + threadIdx.x] = p0;
}

__global__ void update_p(double **p, char **flag, double **rhs, int imax, int jmax, int rb) {
    int i_start, i_end, j_start, j_end;
    init_outer_loop(i_start, i_end);
    init_inner_loop(j_start, j_end);
    for (int i = i_start; i < i_end && i < imax+1; i++) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            if ((i + j) % 2 != rb) { continue; }
            if (flag[i][j] == (C_F | B_NSEW)) {
                /* five point star for interior fluid cells */
                p[i][j] = (1.0 - omega) * p[i][j] - 
                        beta_2 * ((p[i+1][j] + p[i-1][j] ) * rdx2
                                    + (p[i][j+1] + p[i][j-1]) * rdy2
                                    - rhs[i][j]);
            } else if (flag[i][j] & C_F) { 
                /* modified star near boundary */

                double eps_E = ((flag[i+1][j] & C_F) ? 1.0 : 0.0);
                double eps_W = ((flag[i-1][j] & C_F) ? 1.0 : 0.0);
                double eps_N = ((flag[i][j+1] & C_F) ? 1.0 : 0.0);
                double eps_S = ((flag[i][j-1] & C_F) ? 1.0 : 0.0);

                double beta_mod = -omega / ((eps_E + eps_W) * rdx2 + (eps_N + eps_S) * rdy2);
                p[i][j] = (1.0 - omega) * p[i][j] -
                    beta_mod * ((eps_E * p[i+1][j] + eps_W * p[i-1][j]) * rdx2
                                    + (eps_N * p[i][j+1] + eps_S * p[i][j-1]) * rdy2
                                    - rhs[i][j]);
            }
        }
    }

}

__global__ void update_res(double **p, char **flag, double **rhs, int imax, int jmax, double res, double *reduction_buffer) {
    /* computation of residual */
    int i_start, i_end, j_start, j_end;
    init_outer_loop(i_start, i_end);
    init_inner_loop(j_start, j_end);
    for (int i = i_start; i < i_end && i < imax+1; i++) {
        for (int j = j_start; j < j_end && j < jmax+1; j++) {
            if (flag[i][j] & C_F) {
                double eps_E = ((flag[i+1][j] & C_F) ? 1.0 : 0.0);
                double eps_W = ((flag[i-1][j] & C_F) ? 1.0 : 0.0);
                double eps_N = ((flag[i][j+1] & C_F) ? 1.0 : 0.0);
                double eps_S = ((flag[i][j-1] & C_F) ? 1.0 : 0.0);

                /* only fluid cells */
                double add = (eps_E * (p[i+1][j] - p[i][j]) - 
                    eps_W * (p[i][j] - p[i-1][j])) * rdx2  +
                    (eps_N * (p[i][j+1] - p[i][j]) -
                        eps_S * (p[i][j] - p[i][j-1])) * rdy2  -  rhs[i][j];
                res += add * add;
            }
        }
    }
    reduction_buffer[blockIdx.x * blockDim.x + threadIdx.x] = res;
}

/**
 * @brief Red/Black SOR to solve the poisson equation.
 * 
 * @return Calculated residual of the computation
 * 
 */
double poisson() {

    init_p0<<<grid_dim, block_dim>>>(p, flag, imax, jmax, reduction_buffer);
    double p0 = reduce(reduction_buffer);
   
    p0 = sqrt(p0 / fluid_cells); 
    if (p0 < 0.0001) { p0 = 1.0; }

    /* Red/Black SOR-iteration */
    int iter;
    double res = 0.0;
    for (iter = 0; iter < itermax; iter++) {

        for (int rb = 0; rb < 2; rb++) {
            update_p<<<grid_dim, block_dim>>>(p, flag, rhs, imax, jmax, rb);
        }
        update_res<<<grid_dim, block_dim>>>(p, flag, rhs, imax, jmax, res, reduction_buffer);
        res = reduce(reduction_buffer);

        res = sqrt(res / fluid_cells) / p0;

        /* convergence? */
        if (res < eps) break;
    }

    return res;
}


/**
 * @brief Update the velocity values based on the tentative
 * velocity values and the new pressure matrix
 */
__global__ void update_velocity(double **u, double **v, double **p, char ** flag, double **f, double **g, int imax, int jmax, double del_t, double delx, double dely) {
    int i_start, i_end, j_start, j_end;
    init_outer_loop(i_start, i_end);
    init_inner_loop(j_start, j_end);
    for (int i = i_start; i < i_end && i < imax-2; i++) {
        for (int j = j_start; j < j_end && j < jmax-1; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i+1][j] & C_F)) {
                u[i][j] = f[i][j] - (p[i+1][j] - p[i][j]) * del_t / delx;
            }
        }
    }

    for (int i = i_start; i < i_end && i < imax-1; i++) {
        for (int j = j_start; j < j_end && j < jmax-2; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i][j+1] & C_F)) {
                v[i][j] = g[i][j] - (p[i][j+1] - p[i][j]) * del_t / dely;
            }
        }
    }
}


/**
 * @brief Set the timestep size so that we satisfy the Courant-Friedrichs-Lewy
 * conditions. Otherwise the simulation becomes unstable.
 */
void set_timestep_interval() {
    /* del_t satisfying CFL conditions */
    if (tau >= 1.0e-10) { /* else no time stepsize control */
        double umax = 1.0e-10;
        double vmax = 1.0e-10; 
        
        for (int i = 0; i < imax+2; i++) {
            for (int j = 1; j < jmax+2; j++) {
                umax = fmax(fabs(u[i][j]), umax);
            }
        }

        for (int i = 1; i < imax+2; i++) {
            for (int j = 0; j < jmax+2; j++) {
                vmax = fmax(fabs(v[i][j]), vmax);
            }
        }

        double deltu = delx / umax;
        double deltv = dely / vmax; 
        double deltRe = 1.0 / (1.0 / (delx * delx) + 1 / (dely * dely)) * Re / 2.0;

        if (deltu < deltv) {
            del_t = fmin(deltu, deltRe);
        } else {
            del_t = fmin(deltv, deltRe);
        }
        del_t = tau * del_t; /* multiply by safety factor */
    }
}


void main_loop() {
    double res, t;

    apply_boundary_conditions<<<grid_dim, block_dim>>>(u, v, flag, imax, jmax);
    /* Main loop */
    int iters = 0;
    for (t = 0.0; t < t_end; t += del_t, iters++) {
        if (!fixed_dt) {
            set_timestep_interval();
        }

        compute_tentative_velocity<<<grid_dim, block_dim>>>(u, v, flag, f, g, imax, jmax, del_t, Re, delx, dely);
        compute_rhs<<<grid_dim, block_dim>>>(flag, f, g, rhs, imax, jmax, del_t, delx, dely);
        res = poisson();

        update_velocity<<<grid_dim, block_dim>>>(u, v, p, flag, f, g, imax, jmax, del_t, delx, dely);

        apply_boundary_conditions<<<grid_dim, block_dim>>>(u, v, flag, imax, jmax);

        if ((iters % output_freq == 0)) {
            checkCuda(hipDeviceSynchronize());
            printf("Step %8d, Time: %14.8e (del_t: %14.8e), Residual: %14.8e\n", iters, t+del_t, del_t, res);

            if ((!no_output) && (enable_checkpoints)) {
                write_checkpoint(iters, t+del_t);
            }
        }
    } /* End of main loop */

    printf("Step %8d, Time: %14.8e, Residual: %14.8e\n", iters, t, res);
    printf("Simulation complete.\n");

    if (!no_output) {
        write_result(iters, t);
    }
}


/**
 * @brief The main routine that sets up the problem and executes the solving routines routines
 * 
 * @param argc The number of arguments passed to the program
 * @param argv An array of the arguments passed to the program
 * @return int The return value of the application
 */
int main(int argc, char *argv[]) {
    set_defaults();
    parse_args(argc, argv);
    setup();
    checkCuda(hipDeviceSynchronize());

    if (verbose) print_opts();

    allocate_arrays();
    problem_set_up();

    main_loop();


    free_arrays();

    return 0;
}
