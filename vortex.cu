#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <fcntl.h>
#include <math.h>
#include <time.h>

#include "data.cuh"
#include "vtk.cuh"
#include "setup.cuh"
#include "boundary.cuh"
#include "args.cuh"

struct timespec timer;
double get_time() {
	clock_gettime(CLOCK_MONOTONIC, &timer); 
	return (double) (timer.tv_sec + timer.tv_nsec / 1000000000.0);
}

#define time(func, timer) if(print_time){timer = get_time();func;timer = get_time() - timer;}else{func;}

#define print_timer(name, timer) if(print_time)printf("%s: %lf\n", name, timer);

#define init_outer_loop(i, limit, addon) i = threadIdx.x * (imax+2) / blockDim.x;limit = (threadIdx.x+1) * (imax+2) / blockDim.x;if (i == 0) {i = 1;} else if (limit > imax+addon) {i_end = imax+addon;}

#define debug_cuda(i, limit) printf("thread: %d out of %d on block %d. start: %d end: %d\n", threadIdx.x, blockDim.x, blockIdx.x, i, limit);

__device__ double reduce_sum(double value, double *reduction_buffer) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    reduction_buffer[tid] = value;
    __syncthreads();

    if (tid == 0) {
        double sum = 0;
        for (int i = 0; i < gridDim.x * blockDim.x; i++) {
            sum += reduction_buffer[i];
        }
        for (int i = 0; i < gridDim.x * blockDim.x; i++) {
            reduction_buffer[i] = sum;
        }
    }
    __syncthreads();
    return reduction_buffer[tid];
}



/**
 * @brief Computation of tentative velocity field (f, g)
 * 
 */
__device__ void compute_tentative_velocity(double** u, double **v, char **flag, double **f, double **g, int imax, int jmax, double y, double delx, double dely, double del_t) {
    int i, i_end;
    init_outer_loop(i, i_end, 0);
    //debug_cuda(i, i_end);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax+1; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i+1][j] & C_F)) {
                double du2dx = ((u[i][j] + u[i+1][j]) * (u[i][j] + u[i+1][j]) +
                                y * fabs(u[i][j] + u[i+1][j]) * (u[i][j] - u[i+1][j]) -
                                (u[i-1][j] + u[i][j]) * (u[i-1][j] + u[i][j]) -
                                y * fabs(u[i-1][j] + u[i][j]) * (u[i-1][j]-u[i][j]))
                                / (4.0 * delx);
                double duvdy = ((v[i][j] + v[i+1][j]) * (u[i][j] + u[i][j+1]) +
                                y * fabs(v[i][j] + v[i+1][j]) * (u[i][j] - u[i][j+1]) -
                                (v[i][j-1] + v[i+1][j-1]) * (u[i][j-1] + u[i][j]) -
                                y * fabs(v[i][j-1] + v[i+1][j-1]) * (u[i][j-1] - u[i][j]))
                                / (4.0 * dely);
                double laplu = (u[i+1][j] - 2.0 * u[i][j] + u[i-1][j]) / delx / delx +
                                (u[i][j+1] - 2.0 * u[i][j] + u[i][j-1]) / dely / dely;
   
                f[i][j] = u[i][j] + del_t * (laplu / Re - du2dx - duvdy);
            } else {
                f[i][j] = u[i][j];
            }
        }
    }

    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i][j+1] & C_F)) {
                double duvdx = ((u[i][j] + u[i][j+1]) * (v[i][j] + v[i+1][j]) +
                                y * fabs(u[i][j] + u[i][j+1]) * (v[i][j] - v[i+1][j]) -
                                (u[i-1][j] + u[i-1][j+1]) * (v[i-1][j] + v[i][j]) -
                                y * fabs(u[i-1][j] + u[i-1][j+1]) * (v[i-1][j]-v[i][j]))
                                / (4.0 * delx);
                double dv2dy = ((v[i][j] + v[i][j+1]) * (v[i][j] + v[i][j+1]) +
                                y * fabs(v[i][j] + v[i][j+1]) * (v[i][j] - v[i][j+1]) -
                                (v[i][j-1] + v[i][j]) * (v[i][j-1] + v[i][j]) -
                                y * fabs(v[i][j-1] + v[i][j]) * (v[i][j-1] - v[i][j]))
                                / (4.0 * dely);
                double laplv = (v[i+1][j] - 2.0 * v[i][j] + v[i-1][j]) / delx / delx +
                                (v[i][j+1] - 2.0 * v[i][j] + v[i][j-1]) / dely / dely;

                g[i][j] = v[i][j] + del_t * (laplv / Re - duvdx - dv2dy);
            } else {
                g[i][j] = v[i][j];
            }
        }
    }

    /* f & g at external boundaries */
    if (threadIdx.x == 0) {
        for (int j = 1; j < jmax+1; j++) {
            f[0][j]    = u[0][j];
        }
    } else if (threadIdx.x == blockDim.x - 1) {
        for (int j = 1; j < jmax+1; j++) {
            f[imax][j] = u[imax][j];
        }
    }
    init_outer_loop(i, i_end, 1);
    for (i; i < i_end; i++) {
        g[i][0]    = v[i][0];
        g[i][jmax] = v[i][jmax];
    }
}


/**
 * @brief Calculate the right hand side of the pressure equation 
 * 
 */
__device__ void compute_rhs(char **flag, double **f, double **g, double **rhs, int imax, int jmax, double delx, double dely, double del_t) {
    int i, i_end;
    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1;j < jmax+1; j++) {
            if (flag[i][j] & C_F) {
                /* only for fluid and non-surface cells */
                rhs[i][j] = ((f[i][j] - f[i-1][j]) / delx + 
                             (g[i][j] - g[i][j-1]) / dely)
                            / del_t;
            }
        }
    }
}


/**int block_dim = 16;
int grid_dim = 1;
 * @brief Red/Black SOR to solve the poisson equation.
 * 
 * @return Calculated residual of the computation
 * 
 */
__device__ double poisson(double **u, double **v, double **p, char **flag, double **rhs, int imax, int jmax, int fluid_cells, double omega, double beta_2, double rdx2, double rdy2, double *reduction_buffer) {

    double p0 = 0.0;
    /* Calculate sum of squares */
    int i, i_end;
    init_outer_loop(i, i_end, 1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax+1; j++) {
            if (flag[i][j] & C_F) { p0 += p[i][j] * p[i][j]; }
        }
    }
    p0 = reduce_sum(p0, reduction_buffer);
   
    p0 = sqrt(p0 / fluid_cells); 
    if (p0 < 0.0001) { p0 = 1.0; }

    /* Red/Black SOR-iteration */
    int iter;
    double res = 0.0;
    for (iter = 0; iter < itermax; iter++) {

        for (int rb = 0; rb < 2; rb++) {

            init_outer_loop(i, i_end, 1);
            for (; i < i_end; i++) {
                for (int j = 1; j < jmax+1; j++) {
                    if ((i + j) % 2 != rb) { continue; }
                    if (flag[i][j] == (C_F | B_NSEW)) {
                        /* five point star for interior fluid cells */
                        p[i][j] = (1.0 - omega) * p[i][j] - 
                              beta_2 * ((p[i+1][j] + p[i-1][j] ) * rdx2
                                         + (p[i][j+1] + p[i][j-1]) * rdy2
                                         - rhs[i][j]);
                    } else if (flag[i][j] & C_F) { 
                        /* modified star near boundary */

                        double eps_E = ((flag[i+1][j] & C_F) ? 1.0 : 0.0);
                        double eps_W = ((flag[i-1][j] & C_F) ? 1.0 : 0.0);
                        double eps_N = ((flag[i][j+1] & C_F) ? 1.0 : 0.0);
                        double eps_S = ((flag[i][j-1] & C_F) ? 1.0 : 0.0);

                        double beta_mod = -omega / ((eps_E + eps_W) * rdx2 + (eps_N + eps_S) * rdy2);
                        p[i][j] = (1.0 - omega) * p[i][j] -
                            beta_mod * ((eps_E * p[i+1][j] + eps_W * p[i-1][j]) * rdx2
                                         + (eps_N * p[i][j+1] + eps_S * p[i][j-1]) * rdy2
                                         - rhs[i][j]);
                    }
                }
            }
        }
        
        /* computation of residual */
        init_outer_loop(i, i_end, 1);
        for (; i < i_end; i++) {
            for (int j = 1; j < jmax+1; j++) {
                if (flag[i][j] & C_F) {
                    double eps_E = ((flag[i+1][j] & C_F) ? 1.0 : 0.0);
                    double eps_W = ((flag[i-1][j] & C_F) ? 1.0 : 0.0);
                    double eps_N = ((flag[i][j+1] & C_F) ? 1.0 : 0.0);
                    double eps_S = ((flag[i][j-1] & C_F) ? 1.0 : 0.0);

                    /* only fluid cells */
                    double add = (eps_E * (p[i+1][j] - p[i][j]) - 
                        eps_W * (p[i][j] - p[i-1][j])) * rdx2  +
                        (eps_N * (p[i][j+1] - p[i][j]) -
                         eps_S * (p[i][j] - p[i][j-1])) * rdy2  -  rhs[i][j];
                    res += add * add;
                }
            }
        }
        res = reduce_sum(res, reduction_buffer);
        res = sqrt(res / fluid_cells) / p0;
        
        /* convergence? */
        if (res < eps) break;
    }

    return res;
}


/**
 * @brief Update the velocity values based on the tentative
 * velocity values and the new pressure matrix
 */
__device__ void update_velocity(double **u, double **v, double **p, char ** flag, double **f, double **g, int imax, int jmax, double delx, double dely, double del_t) {
    int i, i_end;
    init_outer_loop(i, i_end, -2);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax-1; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i+1][j] & C_F)) {
                u[i][j] = f[i][j] - (p[i+1][j] - p[i][j]) * del_t / delx;
            }
        }
    }
    
    init_outer_loop(i, i_end, -1);
    for (; i < i_end; i++) {
        for (int j = 1; j < jmax-2; j++) {
            /* only if both adjacent cells are fluid cells */
            if ((flag[i][j] & C_F) && (flag[i][j+1] & C_F)) {
                v[i][j] = g[i][j] - (p[i][j+1] - p[i][j]) * del_t / dely;
            }
        }
    }
}


/**
 * @brief Set the timestep size so that we satisfy the Courant-Friedrichs-Lewy
 * conditions. Otherwise the simulation becomes unstable.
 */
void set_timestep_interval() {
    /* del_t satisfying CFL conditions */
    if (tau >= 1.0e-10) { /* else no time stepsize control */
        double umax = 1.0e-10;
        double vmax = 1.0e-10; 
        
        for (int i = 0; i < imax+2; i++) {
            for (int j = 1; j < jmax+2; j++) {
                umax = fmax(fabs(u[i][j]), umax);
            }
        }

        for (int i = 1; i < imax+2; i++) {
            for (int j = 0; j < jmax+2; j++) {
                vmax = fmax(fabs(v[i][j]), vmax);
            }
        }

        double deltu = delx / umax;
        double deltv = dely / vmax; 
        double deltRe = 1.0 / (1.0 / (delx * delx) + 1 / (dely * dely)) * Re / 2.0;

        if (deltu < deltv) {
            del_t = fmin(deltu, deltRe);
        } else {
            del_t = fmin(deltv, deltRe);
        }
        del_t = tau * del_t; /* multiply by safety factor */
    }
}


__global__ void main_loop(double **u, double **v, double **p, char **flag, double **f, double **g, double **rhs, int imax, int jmax, double ui, double vi, double delx, double dely, double del_t, int fluid_cells, double omega, double beta_2, double rdx2, double rdy2, double t_end, int fixed_dt, double y, int output_freq, int no_output, int enable_checkpoints, double *reduction_buffer) {
    double res, t;

	apply_boundary_conditions(u, v, flag, imax, jmax, ui, vi);

    /* Main loop */
    int iters = 0;
    for (t = 0.0; t < t_end; t += del_t, iters++) {
        if (!fixed_dt) {
            //set_timestep_interval();
        }

        compute_tentative_velocity(u, v, flag, f, g, imax, jmax, y, delx, dely, del_t);

        compute_rhs(flag, f, g, rhs, imax, jmax, delx, dely, del_t);

        res = poisson(u, v, p, flag, rhs, imax, jmax, fluid_cells, omega, beta_2, rdx2, rdy2, reduction_buffer);

        update_velocity(u, v, p, flag, f, g, imax, jmax, delx, dely, del_t);

        apply_boundary_conditions(u, v, flag, imax, jmax, ui, vi);

        if ((iters % output_freq == 0) && blockIdx.x == 0 && threadIdx.x == 0) {
            printf("Step %8d, Time: %14.8e (del_t: %14.8e), Residual: %14.8e\n", iters, t+del_t, del_t, res);

            if ((!no_output) && (enable_checkpoints)) {
                //write_checkpoint(iters, t+del_t);
            }
        }
    } /* End of main loop */

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("Step %8d, Time: %14.8e, Residual: %14.8e\n", iters, t, res);
        printf("Simulation complete.\n");
    }
}


/**
 * @brief The main routine that sets up the problem and executes the solving routines routines
 * 
 * @param argc The number of arguments passed to the program
 * @param argv An array of the arguments passed to the program
 * @return int The return value of the application
 */
int main(int argc, char *argv[]) {
    double setup_time;

    setup_time = get_time();
    set_defaults();
    parse_args(argc, argv);
    setup();

    if (verbose) print_opts();

    allocate_arrays();
    problem_set_up();
    setup_time = get_time() - setup_time;
    print_timer("Setup", setup_time);

    main_loop<<<grid_dim, block_dim>>>(u, v, p, flag, f, g, rhs, imax, jmax, ui, vi, delx, dely, del_t, fluid_cells, omega, beta_2, rdx2, rdy2, t_end, fixed_dt, y, output_freq, no_output, enable_checkpoints, reduction_buffer);
    hipDeviceSynchronize();

    if (!no_output) {
        write_result(2000, 5);
    }

    free_arrays();

    return 0;
}
