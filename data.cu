#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>

#include "vtk.cuh"

hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}


__global__ void hello() {
    printf("hello\n");
}

void test() {
	hello<<<1, 1>>>();
    hipDeviceSynchronize();
}

double xlength = 4.0;     /* Width of simulated domain */
double ylength = 1.0;     /* Height of simulated domain */
int imax = 512;           /* Number of cells horizontally */
int jmax = 128;           /* Number of cells vertically */

double t_end = 5.0;        /* Simulation runtime */
double del_t = 0.003;      /* Duration of each timestep */
double tau = 0.5;          /* Safety factor for timestep control */

int itermax = 100;         /* Maximum number of iterations in SOR */
double eps = 0.001;        /* Stopping error threshold for SOR */
double omega = 1.7;        /* Relaxation parameter for SOR */
double y = 0.9;            /* Gamma, Upwind differencing factor in PDE discretisation */

double Re = 500.0;         /* Reynolds number */
double ui = 1.0;           /* Initial X velocity */
double vi = 0.0;           /* Initial Y velocity */

double delx, dely;
double rdx2, rdy2;
double beta_2;

int fluid_cells = 0;

// Grids used for veclocities, pressure, rhs, flag and temporary f and g arrays
int u_size_x, u_size_y;
double ** u;
__device__ double * cuda_u;
int v_size_x, v_size_y;
double ** v;
__device__ double * cuda_v;
int p_size_x, p_size_y;
double ** p;
__device__ double * cuda_p; 
int rhs_size_x, rhs_size_y;
double ** rhs;
__device__ double * cuda_rhs;
int f_size_x, f_size_y;
double ** f;
__device__ double * cuda_f;
int g_size_x, g_size_y;
double ** g;
__device__ double * cuda_g;
int flag_size_x, flag_size_y;
char ** flag;
__device__ char * cuda_flag;

/**
 * @brief Allocate a 2D array that is addressable using square brackets
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return double** A 2D array
 */
double **alloc_2d_array(int m, int n) {
  	double **x;
  	int i;

  	x = (double **)malloc(m*sizeof(double *));
  	x[0] = (double *)calloc(m*n,sizeof(double));
  	for ( i = 1; i < m; i++ )
    	x[i] = &x[0][i*n];
	return x;
}



/**
 * @brief Allocate a 2D char array that is addressable using square brackets
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return char** A 2D array
 */
char **alloc_2d_char_array(int m, int n) {
  	char **x;
  	int i;

  	x = (char **)malloc(m*sizeof(char *));
  	x[0] = (char *)calloc(m*n,sizeof(char));
  	for ( i = 1; i < m; i++ )
    	x[i] = &x[0][i*n];
	return x;
}

double* alloc_2d_cuda_array(int m, int n) {
	size_t pitch;
	double* d_array;

	checkCuda(hipMalloc((void **)&d_array, sizeof(double) * m * n));
	return d_array;
}

char* alloc_2d_char_cuda_array(int m, int n) {
	size_t pitch;
	char* d_array;

	checkCuda(hipMalloc((void **)&d_array, sizeof(char) * m * n));
	return d_array;
}


void to_gpu_2d(void** array, void* cuda_array, int m, int n, int size) {

	//checkCuda(hipMemcpy2D(cuda_array, size, array, size, size, m, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(cuda_array, array[0], size * m * n, hipMemcpyHostToDevice));
}

void from_gpu_2d(void** array, void* cuda_array, int m, int n, int size) {
	checkCuda(hipMemcpy(array[0], cuda_array, size * m * n, hipMemcpyDeviceToHost));
}

/**
 * @brief Free a 2D array
 * 
 * @param array The 2D array to free
 */
void free_2d_array(void ** array) {
	free(array[0]);
	free(array);
}

void free_2d_cuda_array(void *array) {
	checkCuda(hipFree(array));
}
